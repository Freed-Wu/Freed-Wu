#include "hip/hip_runtime.h"
#if 0
bin="$(basename "$0")" && bin="${bin%%.*}" && nvcc -g -G -Xcompiler -Wall -o "$bin" "$0" && exec ./"$bin" "$@" || exit $?
#endif
#include <err.h>
#include <cerrno>
#include <cstdio>
#include <cstdlib>

__global__ void kernel() {
  {% here %}
}

int main(int argc, char *argv[]) {
  kernel<<<1, 1>>>();
  hipDeviceSynchronize();
  return EXIT_SUCCESS;
}
